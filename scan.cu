#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "h_scan.h"
#include "d_scan.h"
#include "wrappers.h"
//config.h defines the number of threads in a block (THREADSPERBLOCK), 
//and the maximum size of shared memory in ints (MAXSHARED) 
#include "config.h"     

//prototypes for functions in this file
void initVector(int * array, int length);
void parseArgs(int argc, char * argv[], int *);
void compare(int * result1, int * result2, int n);
void printUsage();
void printVector(int * vector, int vectorLen);

/*
   driver for the inclusive scan program.  
*/
int main(int argc, char * argv[])
{
    int vectorLen;
    //get the length of the vector
    parseArgs(argc, argv, &vectorLen);
    int * h_vector = (int *) Malloc(sizeof(int) * vectorLen);
    int * h_result = (int *) Malloc(sizeof(int) * vectorLen);
    int * d_result = (int *) Malloc(sizeof(int) * vectorLen);
    float h_time, d_time, speedup;

    //initialize vector 
    initVector(h_vector, vectorLen);
    //printVector(h_vector, vectorLen);
   
    //perform the scan on the CPU
    h_time = h_scan(h_result, h_vector, vectorLen);
    //printVector(h_result, vectorLen);
    printf("\nTiming\n");
    printf("------\n");
    printf("CPU: \t\t%f msec\n", h_time);

    //perform the scan on the GPU 
    d_time = d_scan(d_result, h_vector, vectorLen);
    //printVector(d_result, vectorLen);

    //compare GPU and CPU results 
    compare(h_result, d_result, vectorLen);
    printf("GPU: \t\t%f msec\n", d_time);
    speedup = h_time/d_time;
    printf("Speedup: \t%f\n", speedup);

    free(h_result);
    free(d_result);
    free(h_vector);
}    

/* 
    parseArgs
    This function parses the command line arguments to get
    the vector length.  If the vector length is invalid, 
    it prints usage information and exits.
    Inputs:
    argc - count of the number of command line arguments
    argv - array of command line arguments
    vectorLen - pointer to an int to be set to the vector length
*/
void parseArgs(int argc, char * argv[], int * vectorLen)
{
    int vlen;
    if (argc != 2) printUsage();
    vlen = atoi(argv[1]);        
    if (vlen <= 0 || vlen > (MAXSHARED - THREADSPERBLOCK)) 
       printUsage();
    (*vectorLen) = vlen;
}

/*
    printUsage
    prints usage information and exits
*/
void printUsage()
{
    printf("\nThis program performs an inclusive scan of a vector.\n"); 
    printf("The length of vector to scan is provided as an argument.\n");
    printf("The scan is performed on the CPU and the GPU. The program\n");
    printf("verifies the GPU results by comparing them  to the CPU\n");
    printf("results and outputs the times it takes to perform the scan.\n");
    printf("usage: scan <vector size>\n");
    printf("       <vector size> size of the randomly generated vector to scan\n");
    printf("       <vector size> must not be greater than %d because of the\n", 
                        (MAXSHARED - THREADSPERBLOCK));
    printf("       limited amount of shared memory.\n");
    exit(EXIT_FAILURE);
}

/* 
    initVector
    Initializes an array of ints of size
    length to random values between 0 and 5. 
    Inputs:
    array - pointer to the array to initialize
    length - length of array
*/
void initVector(int * array, int length)
{
    int i;
    for (i = 0; i < length; i++)
    {
        array[i] = (rand() % 5);
    }
}

/*
    compare
    Compares the values in two vectors and outputs an
    error message and exits if the values do not match.
    result1, result2 - int vectors
    n - length of each vector
*/
void compare(int * result1, int * result2, int n)
{
    int i;
    for (i = 0; i < n; i++)
    {
        int diff = abs(result1[i] - result2[i]);
        if (diff != 0) // 
        {
            printf("GPU scan does not match CPU scan.\n");
            printf("cpu result[%d]: %d, gpu: result[%d]: %d\n", 
                   i, result1[i], i, result2[i]);
            exit(EXIT_FAILURE);
        }
    }
}

/*
    printVector
    prints the contents of a vector, 10 elements per line
    vector - pointer to the vector
    vectorLen - length of vector
*/
void printVector(int * vector, int vectorLen)
{
    for (int i = 0; i < vectorLen; i++)
    {
        if ((i % 10) == 0)printf("\n%4d: ", i);
        printf("%3d ", vector[i]);
    }
    printf("\n");
}
