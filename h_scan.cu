#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CHECK.h"
#include "h_scan.h"

//prototype for function local to this file
void scanOnCPU(int * h_result, int * h_vector, int vectorLen);

/*  h_scan
    This function returns the amount of time it takes to perform
    a vector scan on the CPU.
    Inputs:
    h_result - points to the vector to hold the result
    h_vector - points to the vector to scan
    vectorLen - length of the vector to scan

    returns the amount of time it takes to perform the
    convolution
*/
float h_scan(int * h_result, int * h_vector, int vectorLen)
{
    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use CUDA functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_cpu));  
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));   
    
    //call function that does the actual work
    scanOnCPU(h_result, h_vector, vectorLen);
   
    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu)); 

    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

/*  h_scan
    This function performs the vector scan on the CPU.  
    Inputs:
    h_result - points to the vector to hold the result
    h_vector - points to the vector to scan
    vectorLen - length of the vector to scan

    modifies the h_result vector
*/
void scanOnCPU(int * h_result, int * h_vector, int vectorLen)
{
    int i, accumulator = 0;

    for (i = 0; i < vectorLen; i++)
    {
        accumulator += h_vector[i];
        h_result[i] = accumulator;
    }
}
